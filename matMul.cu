#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "jh_timer.h"

#define BLOCK_SIZE 16

#define SIZE_M 1024
#define SIZE_N 2048
#define SIZE_K 1024

__global__ void matMul(int* matA, int* matB, int* matC, int m, int n, int k);

int main(void)
{
    // set matrix size
    int m = SIZE_M;
    int n = SIZE_N;
    int k = SIZE_K;
    printf("Matrix size: A(%d, %d), B(%d, %d), C(%d, %d)\n", m, k, k, n, m, n);

    int sizeA = m * k;
    int sizeB = k * n;
    int sizeC = m * n;

    // initialize timer
    JH_timer timer = timer_init(4);

    // host memory allocation
    int* A = NULL;
    int* B = NULL;
    int* C = NULL;
    
    A = (int*)malloc(sizeof(int) * sizeA);
    B = (int*)malloc(sizeof(int) * sizeB);
    C = (int*)malloc(sizeof(int) * sizeC);

    // generate input matrices
	for (int i = 0; i < sizeA; i++) A[i] = ((rand() % 10) + ((rand() % 100) / 100.0));
	for (int i = 0; i < sizeB; i++) B[i] = ((rand() % 10) + ((rand() % 100) / 100.0));

    // device memory allocation
    int *dA, *dB, *dC;
    hipMalloc(&dA, sizeof(int) * sizeA);
    hipMalloc(&dB, sizeof(int) * sizeB);
    hipMalloc(&dC, sizeof(int) * sizeC);

    setTimerName(&timer, 0, "matMul (GPU) total");
    onTimer(&timer, 0);

    // host -> device memory transfer
    setTimerName(&timer, 1, "Host -> Device memory transfer");
    onTimer(&timer, 1);
    hipMemcpy(dA, A, sizeof(int) * sizeA, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(int) * sizeB, hipMemcpyHostToDevice);
    offTimer(&timer, 1);

    // matrix multiplication on device (GPU)
    dim3 gridDim(ceil((float)n / BLOCK_SIZE), ceil((float)m / BLOCK_SIZE));
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    printf("Grid shape: (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
    printf("Block shape: (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);

    setTimerName(&timer, 2, "matMul (GPU)");
    onTimer(&timer, 2);
    matMul<<<gridDim, blockDim>>>(dA, dB, dC, m, n, k);
    hipDeviceSynchronize();
    printf("matMul (GPU) ended.\n");
    offTimer(&timer, 2);

    // device -> host memory transfer
    setTimerName(&timer, 3, "Device -> Host memory transfer");
    onTimer(&timer, 3);
    hipMemcpy(C, dC, sizeof(int) * sizeC, hipMemcpyDeviceToHost);
    offTimer(&timer, 3);

    offTimer(&timer, 0);

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // print log to stdout
    printLog(&timer, NULL);

    free(A);
    free(B);
    free(C);

    return 0;
}

__global__ void matMul(int* matA, int* matB, int* matC, int m, int n, int k)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // If the index is out of range, do nothing
    if (row >= m || col >= n) return;

    int index = row * n + col;
    matC[index] = 0;

    for (int p = 0; p < k; p++)
        matC[index] += matA[row * k + p] * matB[p * n + col];
}
